#include <shared/cuda_helpers.cuh>

namespace ppt
{
namespace shared
{

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";

        std::cerr << "this here: " << hipGetErrorString(result) << std::endl;
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

} // namespace shared
} // namespace ppt